#include "hip/hip_runtime.h"
#include "kernels.h"

__global__ void assign_order_kernel(int *object){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	object[id] = id;
}

void assign_order(int *objects, int gridSize, int blockSize, hipStream_t stream){
	assign_order_kernel<<<gridSize, blockSize, 0, stream>>>(objects);
}


